#include "hip/hip_runtime.h"
#include <stdexcept>
#include "ops/kernels/fused_op.cuh"
#include "cuda_defines.cuh"
#include "type_traits/types.cuh"

namespace om {

    
    template <typename T, typename Op>
    __global__ void apply_op_rank1(const DeviceTensorView<const T> src, DeviceTensorView<T> dst, Op op) {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        if (x < dst.shape[0])
            dst(x) = op(src(x));
    }
    
    template <typename T, typename Op>
    __global__ void apply_op_rank2(const DeviceTensorView<const T> src, DeviceTensorView<T> dst, Op op) {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;
        
        if (y < dst.shape[0] && x < dst.shape[1])
            dst(y, x) = op(src(y, x));
    }
    
    template <typename T, typename Op>
    __global__ void apply_op_rank3(const DeviceTensorView<const T> src, DeviceTensorView<T> dst, Op op) {
        size_t x = blockIdx.x * blockDim.x + threadIdx.x;
        size_t y = blockIdx.y * blockDim.y + threadIdx.y;
        size_t z = blockIdx.z * blockDim.z + threadIdx.z;
        
        if (z < dst.shape[0] && y < dst.shape[1] && x < dst.shape[2])
            dst(z, y, x) = op(src(z, y, x));
    }

    template <typename T, typename Op>
    __global__ void apply_op_rank4(const DeviceTensorView<const T> src, DeviceTensorView<T> dst, Op op) {
        size_t w = threadIdx.x + blockIdx.x * blockDim.x;
        size_t h = threadIdx.y + blockIdx.y * blockDim.y;
        size_t n = blockIdx.z; // N dimension
    
        for (size_t c = threadIdx.z; c < dst.shape[1]; c += blockDim.z) {
            if (n < dst.shape[0] &&
                c < dst.shape[1] &&
                h < dst.shape[2] &&
                w < dst.shape[3]) {
                dst(n, c, h, w) = op(src(n, c, h, w));
            }
        }
    }
    
    template <typename T, typename Op>
    __global__ void apply_op_nd(const DeviceTensorView<const T> src, DeviceTensorView<T> dst, Op op) {
        size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t total_elements = dst.size();
    
        if (idx >= total_elements) return;
    
        size_t offset = 0;
        size_t tmp = idx;
        for (size_t d = 0; d < dst.rank; ++d) {
            size_t coord = tmp % dst.shape[d];
            offset += coord * dst.stride[d];
            tmp /= dst.shape[d];
        }
    
        dst[offset] = op(src[offset]);
    }

    template <typename T, typename Op>
    void launch_apply_op(const TensorView<const T> src, TensorView<T> dst, Op op)
    {
        if (!dst.match(src))
            throw std::invalid_argument("Source and destination must have the same shape");

        switch (dst.rank)
        {
        case 1:
            {
                dim3 threads(16);
                dim3 blocks((dst.shape[0] + 15) / 16);
                apply_op_rank1<<<blocks, threads>>>(src.as_device_tw(), dst.as_device_tw(), op);
            }
            break;
        case 2:
            {
                dim3 threads(16, 16);
                dim3 blocks((dst.shape[1] + 15) / 16, (dst.shape[0] + 15) / 16);
                apply_op_rank2<<<blocks, threads>>>(src.as_device_tw(), dst.as_device_tw(), op);
            }
            break;
        case 3:
            {
                dim3 threads(8, 8, 8); // 512 threads per block
                dim3 blocks((dst.shape[2] + 7) / 8, (dst.shape[1] + 7) / 8, (dst.shape[0] + 7) / 8);
                apply_op_rank3<<<blocks, threads>>>(src.as_device_tw(), dst.as_device_tw(), op);
            }
            break;
        case 4:
            {
                dim3 threads(8, 8, dst.shape[1] < 8 ? dst.shape[1] : 8); // avoid large z
                dim3 blocks(
                    (dst.shape[3] + threads.x - 1) / threads.x,
                    (dst.shape[2] + threads.y - 1) / threads.y,
                    dst.shape[0]
                );
                apply_op_rank4<<<blocks, threads>>>(src.as_device_tw(), dst.as_device_tw(), op);
            }
            break;
        default:
            {
                size_t total_elements = dst.size();
                dim3 threads(256);
                dim3 blocks((total_elements + threads.x - 1) / threads.x);
                apply_op_nd<<<blocks, threads>>>(src.as_device_tw(), dst.as_device_tw(), op);
            }
            break;
        }
        CUDA_CHECK;
        hipDeviceSynchronize();
    }

    // Explicit instantiations
    template void launch_apply_op<float>(const TensorView<const float> src, TensorView<float> dst, Add<float> op);
    template void launch_apply_op<int>(const TensorView<const int> src, TensorView<int> dst, Add<int> op);
    template void launch_apply_op<char>(const TensorView<const char> src, TensorView<char> dst, Add<char> op);
    template void launch_apply_op<float16_t>(const TensorView<const float16_t> src, TensorView<float16_t> dst, Add<float16_t> op);
}

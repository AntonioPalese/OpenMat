#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>


#define CUDA_CHECK(err) do { cuda_check((err), __FILE__, __LINE__); } while(false)
inline void cuda_check(hipError_t error_code, const char *file, int line)
{
    if (error_code != hipSuccess)
    {
        fprintf(stderr, "CUDA Error %d: %s. In file '%s' on line %d\n", error_code, hipGetErrorString(error_code), file, line);
        fflush(stderr);
        exit(error_code);
    }
}

// Distributed Shared memory histogram kernel
__global__ void clusterHist_kernel(int *bins, const int nbins, const int bins_per_block, const int *__restrict__ input,
                                   size_t array_size)
{
  extern __shared__ int smem[];
  namespace cg = cooperative_groups;
  int tid = cg::this_grid().thread_rank();

  // Cluster initialization, size and calculating local bin offsets.
  cg::cluster_group cluster = cg::this_cluster();
  unsigned int clusterBlockRank = cluster.block_rank();
  int cluster_size = cluster.dim_blocks().x;

  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    smem[i] = 0; //Initialize shared memory histogram to zeros
  }

  // cluster synchronization ensures that shared memory is initialized to zero in
  // all thread blocks in the cluster. It also ensures that all thread blocks
  // have started executing and they exist concurrently.
  cluster.sync();

  for (int i = tid; i < array_size; i += blockDim.x * gridDim.x)
  {
    int ldata = input[i];

    //Find the right histogram bin.
    int binid = ldata;
    if (ldata < 0)
      binid = 0;
    else if (ldata >= nbins)
      binid = nbins - 1;

    //Find destination block rank and offset for computing
    //distributed shared memory histogram
    int dst_block_rank = (int)(binid / bins_per_block);
    int dst_offset = binid % bins_per_block;

    //Pointer to target block shared memory
    int *dst_smem = cluster.map_shared_rank(smem, dst_block_rank);

    //Perform atomic update of the histogram bin
    atomicAdd(dst_smem + dst_offset, 1);
  }

  // cluster synchronization is required to ensure all distributed shared
  // memory operations are completed and no thread block exits while
  // other thread blocks are still accessing distributed shared memory
  cluster.sync();

  // Perform global memory histogram, using the local distributed memory histogram
  int *lbins = bins + cluster.block_rank() * bins_per_block;
  for (int i = threadIdx.x; i < bins_per_block; i += blockDim.x)
  {
    atomicAdd(&lbins[i], smem[i]);
  }
}

int main(void)
{

  int array_size = 1024;
  int threads_per_block = 1024;
  int nbins = 20;  
  int* bins = NULL;
  const int* __restrict__ input = NULL;


  hipLaunchConfig_t config = {0};
  config.gridDim = array_size / threads_per_block;
  config.blockDim = threads_per_block;

  // cluster_size depends on the histogram size.
  // ( cluster_size == 1 ) implies no distributed shared memory, just thread block local shared memory
  int cluster_size = 2; // size 2 is an example here
  int nbins_per_block = nbins / cluster_size;

  //dynamic shared memory size is per block.
  //Distributed shared memory size =  cluster_size * nbins_per_block * sizeof(int)
  config.dynamicSmemBytes = nbins_per_block * sizeof(int);

  CUDA_CHECK(::hipFuncSetAttribute((void *)clusterHist_kernel, hipFuncAttributeMaxDynamicSharedMemorySize, config.dynamicSmemBytes));

  hipLaunchAttribute attribute[1];
  attribute[0].id = cudaLaunchAttributeClusterDimension;
  attribute[0].val.clusterDim.x = cluster_size;
  attribute[0].val.clusterDim.y = 1;
  attribute[0].val.clusterDim.z = 1;

  config.numAttrs = 1;
  config.attrs = attribute;

  cudaLaunchKernelEx(&config, clusterHist_kernel, bins, nbins, nbins_per_block, input, array_size);
}
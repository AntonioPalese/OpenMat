#include "hip/hip_runtime.h"
#include "cudautils.h"
#include <stdio.h>

__global__ void _print(float* A, size_t r, size_t c)
{
    for(int i = 0; i < r; i++)
    {
        for(int j = 0; j < c; j++)
        {
            printf("%f ", A[j+i*c]);
        }
        printf("\n");
    }
}

void print_cuda_err(hipError_t err, const char* tag)
{
    if(err != hipSuccess)
        printf("[CUDA ERROR (%s)] :  %s\n", tag, hipGetErrorString(err));
}

void cuda_print(float* mat, size_t r, size_t c)
{
    _print<<<1, 1>>>(mat, r, c);
}

void print(float* mat, size_t r, size_t c)
{
    for(int i = 0; i < r; i++)
    {
        for(int j = 0; j < c; j++)
        {
            printf("%f ", mat[j+c*i]);
        }
        printf("\n");
    }
}

void to_device(Matrix* dst, const Matrix* src)
{
    dst->cols = src->cols;
    dst->rows = src->rows;
    dst->cuda = true;
    dst->stride = src->stride;

    hipDeviceSynchronize();
    size_t size = src->rows*src->cols*sizeof(float);
    hipError_t err = hipMalloc((void**)&dst->data, size);
    print_cuda_err(err, "hipMalloc");
    err = hipMemcpy(dst->data, src->data, size, hipMemcpyHostToDevice);
    print_cuda_err(err, "hipMemcpy");
}

void to_host(Matrix* dst, const Matrix* src)
{
    hipDeviceSynchronize();
    size_t size = src->rows*src->cols*sizeof(float);
    hipError_t err = hipMemcpy(dst->data, src->data, size, hipMemcpyDeviceToHost);
    print_cuda_err(err, "hipMemcpy");
}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>
#include <stdlib.h>

//#define STRCAT(dst, src) strcat(dst, src)
#define CUDA_CHECK(op) cuda_check(op,#op)
void cuda_check(hipError_t err, const char* desc)
{
    if(err != hipError_t::hipSuccess)
    {
        printf("[CUDA ERROR ( %s )] : %s\n", desc, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__global__ void hostMempryAccessingKernel(float *dptr)
{
    *dptr = 1.0f;
    printf("dptr in kernel : %f\n", *dptr);
}

int main(void)
{
    size_t n = 1;
    size_t size = n * sizeof(float);


    float* h_a;
    
    CUDA_CHECK(hipHostAlloc(&h_a, size, hipHostMallocMapped));
    float* dprt;
    CUDA_CHECK(hipHostGetDevicePointer(&dprt, h_a, 0));
    hostMempryAccessingKernel<<<1, 1>>>(dprt);
    CUDA_CHECK(hipGetLastError());

    // the kernel is executed asynch, even before the printf buffer could be loaded
    CUDA_CHECK(hipDeviceSynchronize());

    printf("value of the pointer accessed by host after kernal execution : %f\n", *h_a);
}

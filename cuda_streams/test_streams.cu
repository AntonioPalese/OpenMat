
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string>

#define CUDA_CHECK(op) cuda_check(op, #op);
void cuda_check(hipError_t err, const char* desc)
{
    if(err != hipError_t::hipSuccess)
    {
        printf("[CUDA ERROR ( %s )] : %s\n", desc, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// -------------------------------------------------------------------------------------------
// -------------------------------------------------------------------------------------------
// -------------------------------------------------------------------------------------------


__global__ void kernelTest()
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("idx : %d\n", idx);
}

int main(void)
{
    size_t n = pow(2, 28);
    size_t size = n*sizeof(float);

    float* h_a, * h_b;
    float* d_a;

    

    CUDA_CHECK(hipHostMalloc((void**)&h_a, size, hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc((void**)&h_b, size, hipHostMallocDefault));
    CUDA_CHECK(hipMalloc((void**)&d_a, size));
    //h_a = (float*) malloc(size);
    //h_b = (float*) malloc(size);    

    memset(h_a, 0xffffffff, size);

    /////////    

    hipStream_t streams[2];
    for(int i = 0; i < 2; i++)
    {
         CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    for(int i = 0; i < 2; i++)
    {
        int half_size = size/2;
        int half_elements = n/2;
        CUDA_CHECK(hipMemcpyAsync(d_a + i * half_elements, h_a + i * half_elements, half_size, hipMemcpyHostToDevice, streams[i]));  

        dim3 blockDim(1024);
        dim3 gridDim(n / 1024);
        kernelTest<<<gridDim, blockDim, 0, streams[i]>>>();
        CUDA_CHECK(hipGetLastError());  

        CUDA_CHECK(hipMemcpyAsync(h_b + i * half_elements, d_a + i * half_elements, half_size, hipMemcpyDeviceToHost, streams[i]));
        //CUDA_CHECK(cudaStreamSynchronize(streams[i]));
    }

    for(int i = 0; i < 2; i++)
    {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }


    /////////

    // CUDA_CHECK(cudaMemcpy(d_a, h_a, size, cudaMemcpyHostToDevice));

    // kernelTest<<<1, n>>>();
    // CUDA_CHECK(cudaGetLastError());

    // CUDA_CHECK(cudaMemcpy(h_b, d_a, size, cudaMemcpyDeviceToHost ));

    CUDA_CHECK(hipFree(d_a));
    hipHostFree(h_a);
    hipHostFree(h_b);
    // free(h_a);
    // free(h_b);

}